//=============================================================//
//                                                             //
//                      || CUDA-Q ||                           //
//                                                             //
//              --------Version 1.0----------                  //
//                                                             //
//                                                             //
//                                                             //
//    Authors: Andrew Konya      (Kent State University)       //
//             Jonathan Selinger (Kent State University)       // 
//                                                             //
//=============================================================//


#include "mainhead.h"






int main()
{

	
	//Get Device properties
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop,0));
	printf( "Code executing on %s\n\n", prop.name );
	//displayGPUinfo(prop);

	int Ntets,Nnodes;
	//get dimensions of the mesh
	get_mesh_dim(Ntets,Nnodes);
	
	//create objects of TetArray and NodeArray class with correct size
	TetArray Tet = TetArray(Ntets);
	NodeArray Node = NodeArray(Nnodes);

	//read the mesh into Node and Tet objects
	get_mesh(Node,Tet,Ntets,Nnodes);

	//get positions of tetrahedra
	get_tet_pos(Node,Tet,Ntets);

	//reorder tetrahedra 
	gorder_tet(Node,Tet,Ntets);

	//re-order nodes and reassing tetrahedra component lists
	finish_order(Node,Tet,Ntets,Nnodes);

	//find initial A's and invert them  store all in Tet object
	init_As(Node,Tet,Ntets);

	//now ready to prepare for dyanmics
	//delcare data stuctures for data on device
	//and host
	DevDataBlock dev_dat;
	HostDataBlock host_dat;

	//Pack data to send to device
	packdata(Node,Tet,&host_dat,Ntets,Nnodes);

	//send data to device
	data_to_device(&dev_dat,&host_dat,Ntets,Nnodes);


	//=================================================================
	//run dynamics
	//=================================================================

	//run_dynamics(&dev_dat,&host_dat,Ntets,Nnodes);

	//check for CUDA erros
	any_errors();

	//exit program
	exit_program(&dev_dat);

  return 0;
}
